#include "hip/hip_runtime.h"
/*
 *  dmm_gpu.cu -- Template for DMM GPU kernels
 *
 *  Copyright (C) 2020, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2020, Athena Elafrou/Petros Anastasiadis
 */
#include "dmm.h"
#include "stdio.h"
#include <hipblas.h>

/*
 *  Naive kernel
 */
__global__ void dmm_gpu_naive(const value_t *A, const value_t *B, value_t *C,
		const size_t M, const size_t N, const size_t K) {

	/* Compute the row and the column of the current thread */

	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int row = by*blockDim.y + ty;
	int col = bx*blockDim.x + tx;

	value_t dot = 0;

	/* If the thread's position is out of the array, it remains inactive */
	while (row < M && col < N){
		/* Compute the value of C */
		for (int k = 0; k < K; k++){
			dot += A[row*K+k]*B[col+k*N];
		}
		C[row*N+col]=dot;
		row += blockDim.y*gridDim.y;
		col += blockDim.x*gridDim.x;
	}

}

/*
 *  Coalesced memory acceses of A.
 */
__global__ void dmm_gpu_coalesced_A(const value_t *A, const value_t *B,
		value_t *C, const size_t M, const size_t N,
		const size_t K) {

	/* Define the shared memory between the threads of the same thread block */
	__shared__ value_t A_shared[TILE_Y*TILE_X];

	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	/* Compute the tile of the current thread */
	int row = by * TILE_Y + ty;
	int col = bx * TILE_X + tx;

	value_t dot = 0;

	for(int m = 0; m < K; m+=TILE_X){
		/* Load the current tile in the shared memory and synchronize */
		A_shared[ty*TILE_X+tx] = A[row*K + m+tx];

		__syncthreads();

		for(int k = 0; k < TILE_X; k++){
			/* Compute the inner product of current tile and synchronize */
			// This has to be A goes from tile to tile 
			// But B traverse the hole block so is the same with 
			// the naive but you have to change from B[k*N + col]
			// to B[(m+k)*N+col] because here k is only in the tile
			dot += A_shared[ty*TILE_X+k]*B[(m+k)*N+col];
		}
		__syncthreads();
	}
	/* Save result */
	C[row*N+col] = dot;

}

/*
 *  Reduced memory accesses.
 */
__global__ void dmm_gpu_reduced_global(const value_t *A, const value_t *B, value_t *C,
		const size_t M, const size_t N, const size_t K) {

	/* Define the shared memory between the threads of the same thread block */
	__shared__ value_t A_shared[TILE_Y*TILE_X];
	__shared__ value_t B_shared[TILE_Y*TILE_X];

	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int row = by * TILE_Y + ty;
	int col = bx * TILE_X + tx;

	value_t dot = 0;

	for(int m = 0; m < K; m+=TILE_X){
		A_shared[ty*TILE_X+tx] = A[row*K + m+tx];
		// If TILE.x != TILE.y we have to do this differently
		// I dont think this can happen because we add more 
		// complexity than nessesary. Either way every element will  
		// be excecuted in 32 wraps so no need for extra thinking.
		B_shared[ty*TILE_X+tx] = B[col + (m+ty)*N];
		
		__syncthreads();

		for(int k = 0; k < TILE_X; k++){
			dot += A_shared[ty*TILE_X+k]*B_shared[k*TILE_X+tx];
		}
		__syncthreads();
	}
	C[row*N+col] = dot;

}

/*
 *  Use of cuBLAS
 */
// C(m,n) = A(m,k) * B(k,n)
void dmm_gpu_cublas(const value_t *A, const value_t *B, value_t *C,
		const size_t M, const size_t N, const size_t K) {

	/* Define parameters for hipblasSgemm */

	int lda = N;
	int ldb = K;
	int ldc = N;
	
	const float alph = 1;
	const float bet = 0;
	const float *alpha = &alph;
	const float *beta = &bet;
	
	/* Create a handle for CUBLAS */
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	/* Compute the matrix multiplication */
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, alpha, A, lda, B, ldb, beta, C, ldc);

	/* Destroy the handle */
	hipblasDestroy(handle);
}
